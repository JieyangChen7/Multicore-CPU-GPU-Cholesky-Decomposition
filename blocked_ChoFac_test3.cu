#include "hip/hip_runtime.h"
/*Blocked Cholesky Factorization v1.4.
*potf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
*Jieyang Chen, University of California, Riverside
**/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing

#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"cblas.h"
#include<vector>
//#include"lapacke.h"
//#include"blas.h"
//#include<hip/hip_runtime.h>
//#include<hip/hip_runtime.h>

#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;

void printMatrix_host(double * matrix_host, int N);
void printMatrix_gpu(double * matrix_device, size_t matrix_pitch, int N);
void POTF2_CPU(char uplo, double * matrix, int ld, int B);
__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta);
void matrixGenerator_gpu(char uplo, double * matrix, int matrix_ld, double * result, int result_ld, int N,  int B);
__global__ void resultVerify_gpu_help(double * realResult,int real_ld, double * testResult,int test_ld,double * diff, int N);
bool resultVerify_gpu(double * realResult,int real_ld, double * testResult, int test_ld, int N, int B);
void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,float * real_time, float * proc_time, long long * flpins, float * mflops);


long long int SYRK_Flops(long long N, long long B, long long I){
    long long int flop_num = 0;
    //int b_size;
    //for(int i=B;i<N;i+=B){
        //b_size = min(B,N-i);
        //flop_num += 2 * b_size * b_size * i + b_size*b_size;
    
    //}
    //return flop_num;
    if ( N > 0 && I > 0 && I < N ) {
        
        flop_num = 2 * B * B * I + B*B;
        if (flop_num<0) {
            cout<<"ERROR:syrk"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}

long long GEMM_Flops(long long N, long long B, long long I){
    long long flop_num = 0;
   
    if ( N > 0 && I > 0 && N-I-B > 0 ) {
        flop_num = 2 * B * (N-I-B) * I + B*(N-I-B);
        if (flop_num<0) {
            cout<<"ERROR:gemm"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}
        
long long TRSM_Flops(long long N, long long B, long long I){
    long long flop_num = 0;
    /*int b_size;
    for(int i=0;i<N-B;i+=B){
        b_size = min(B,N-i);
        flop_num += b_size * b_size * (N-i-b_size);
    }
    return flop_num;
     */
    if (N>0&&(!(I<0))&&N-I-B>0) {
        flop_num = B * B * (N-I-B);
        if (flop_num<0) {
            cout<<"ERROR:trsm"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}

long POTRF_Flops(int B){
    long long flop_num = 0;
    int b_size = B;
    if(B>64){
        for (int i=0; i<B; i+=64) {
            b_size = min(64, B-i);
            //SYRK
            if(i>0){
                flop_num += SYRK_Flops(B, b_size, i);
            }
            //GEMM
            if(i>0&&i<B-b_size){
                flop_num += GEMM_Flops(B, b_size, i);
            }
            //POTRF2
            for(int j=0;j<b_size;j++){
                flop_num += 2*j + 1 + (b_size-j-1)*i + b_size-j-1;
            }
            //TRSM
            flop_num += TRSM_Flops(B, b_size, i);
        }
    }else{
      for(int i=0;i<B;i++){
          flop_num += 2*i + 1 + (B-i-1)*i + B-i-1;
      }
    }
    if (flop_num<0) {
        cout<<"ERROR:potrf  "<<flop_num<<"  "<<B<<" "<<endl;
    }
    return flop_num;
}

double COPY_time(int B){
    
    float real_time = 0.0;
    float proc_time = 0.0;
    long long flpins = 0.0;
    float mflops = 0.0;
    
    
    float total_real_time = 0.0;
    float total_proc_time = 0.0;
    long long total_flpins = 0.0;
    float total_mflops = 0.0;
    
    int TEST_NUM = 10;
    
    char uplo = 'u';
    double * matrix;
    double * result;
    double * temp;
    
    size_t matrix_pitch;
    size_t result_pitch;
    //Memory allocation on RAM and DRAM
    hipMallocPitch((void**)&matrix,&matrix_pitch,B*sizeof(double),B);
    hipMallocPitch((void**)&result,&result_pitch,B*sizeof(double),B);
    hipHostAlloc((void**)&temp,B*B*sizeof(double),hipHostMallocDefault);
    
    int matrix_ld= matrix_pitch/sizeof(double);
    int result_ld= result_pitch/sizeof(double);
    
    matrixGenerator_gpu(uplo,matrix,matrix_ld,result,result_ld,B,2);
    
    
    for(int i=0;i<TEST_NUM;i++){
    
      if(PAPI_flops( &real_time, &proc_time, &flpins, &mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return -1;
      }
      hipMemcpy2D(temp,B*sizeof(double),matrix,matrix_pitch,B*sizeof(double),B,hipMemcpyDeviceToHost);
      hipMemcpy2D(matrix,matrix_pitch,temp,B*sizeof(double),B*sizeof(double),B,hipMemcpyHostToDevice);
      if(PAPI_flops( &real_time, &proc_time, &flpins, &mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return -1;
      }
      PAPI_shutdown();
      total_real_time += real_time;
      total_proc_time += proc_time;
    }
    

    
    
    
    hipHostFree(temp);
    hipFree(matrix);
    hipFree(result);
    
    double time =(total_real_time/(double)TEST_NUM);
    if (time<0) {
        cout<<"ERROR:copy"<<B<<" "<<endl;
    }
    return time;

}

int currentOptimalB(int N, int i, int G){
    long long GPUflops = 300000000000;
    long long CPUflops = 6500000000;
    int bestB = 2;
    double bestT = 10000000;

    for(int B=1;B<(N-i)/2;B+=G){
      double current_time = 0;
      if (i>0) {
	current_time += (double)SYRK_Flops(N,B,i)/GPUflops;
      }
      if(i>0 && i<N-B){
	current_time += (double)GEMM_Flops(N,B,i)/GPUflops;
      }
      double t1 = (double)COPY_time(B) + (double)POTRF_Flops(B)/CPUflops;
      double t2 = 0;
      if(i>0 && i<N-B){
	t2 += (double)GEMM_Flops(N,B,i)/GPUflops;
      }
      current_time += max(t1,t2);
      if(i<N-B){
	current_time += (double)TRSM_Flops(N,B,i)/GPUflops;
      }
      if(current_time < bestT){
	bestT = current_time;
	bestB = B;
      }
    }
    cout<<"i="<<i<<"   B:"<<bestB<<endl;
    return bestB;
}

vector<int> optimalB(int N, int G){
  int b_size = 0;
  vector<int> block_sizes;
  for(int i=0;i<N;i+=b_size){
    b_size = currentOptimalB(N, i, G);
    b_size = min(b_size,N-i);
    block_sizes.push_back(b_size);
  }
  return block_sizes;
}


void printMatrix_host(double * matrix_host, int N){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            cout.width(5);
            cout.setf(ios::left);
            cout<<matrix_host[j*N+i];
        }
        cout<<endl;
    }
    cout<<endl;
}

void printMatrix_gpu(double * matrix_device, size_t matrix_pitch, int N){
    double * matrix_host = new double[N*N]();
    hipMemcpy2D(matrix_host,N*sizeof(double),matrix_device,matrix_pitch,N*sizeof(double),N,hipMemcpyDeviceToHost);
    printMatrix_host(matrix_host,N);
    delete[] matrix_host;
}

//cpu version
void POTF2_CPU(char uplo, double * matrix, int ld, int B){
  if(uplo == 'u'){
    for(int i = 0; i<B;i++){
      matrix[i*ld+i] = sqrt(matrix[i*ld+i]);
      for(int j=i+1;j<B;j++){
        matrix[j*ld+i] /=matrix[i*ld+i];
      }
      for(int j=i+1;j<B;j++){
        for(int k=i+1;k<j+1;k++){
          matrix[j*ld+k]-=matrix[j*ld+i]*matrix[k*ld+i];
        }
      }
    }
  } 
  if(uplo == 'l'){
    for(int i = 0; i<B;i++){
      matrix[i*ld+i] = sqrt(matrix[i*ld+i]);
      for(int j=i+1;j<B;j++){
	matrix[i*ld+j] /=matrix[i*ld+i];
      }
      for(int j=i+1;j<B;j++){
	for(int k=i+1;k<j+1;k++){
	  matrix[k*ld+j]-=matrix[i*ld+j]*matrix[i*ld+k];
	}
      }
    }
  }
}



__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta){
    int col = threadIdx.x+blockIdx.x*blockDim.x;
    int row = threadIdx.y+blockIdx.y*blockDim.y;
    if(uplo == 'u'){
        if(row<col+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    }
    else{
        if(col<row+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    } 
}


void matrixGenerator_gpu(char uplo, double * matrix, int matrix_ld, double * result, int result_ld, int N,  int B){
    double a = 10.0;
    //initialize cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    //initialize hiprand
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,10ULL);
    //generate random number in range (0,1] on result using hiprand
    hiprandGenerateUniformDouble(gen,result,result_ld*N);
    hipDeviceSynchronize();
    //printMatrix_gpu(result,result_ld*sizeof(double),N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(result, result_ld, uplo, a,1);
    hipDeviceSynchronize();
    //printMatrix_gpu(result,N);
    //do matrix-matrix multiplcation using cublas
    hipMemset(matrix,0,matrix_ld*N*sizeof(double));
    
    
    double alpha = 1.0;
    double beta = 1.0;
    if(uplo == 'u'){
        hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,N,N,N,&alpha,result,result_ld,result,result_ld,&beta,matrix,matrix_ld);
    }
    else if(uplo == 'l'){
        hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,N,N,&alpha,result,result_ld,result,result_ld,&beta,matrix,matrix_ld);
    }
    hipDeviceSynchronize();
    //printMatrix_gpu(matrix,N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(matrix, matrix_ld, uplo, 1.0,0);
    hipDeviceSynchronize();
    //printMatrix_gpu(matrix,matrix_ld*sizeof(double), N);
}
 

__global__ void resultVerify_gpu_help(double * realResult,int real_ld, double * testResult,int test_ld,double * diff, int N){
    int col = threadIdx.x+blockIdx.x*blockDim.x;
    int row = threadIdx.y+blockIdx.y*blockDim.y;
    diff[col*N+row] = testResult[col*test_ld+row] - realResult[col*real_ld+row];
}

bool resultVerify_gpu(double * realResult,int real_ld, double * testResult, int test_ld, int N, int B){
    double * diff;
    hipMalloc((void**)&diff,N*N*sizeof(double));
    resultVerify_gpu_help<<<dim3(N/B,N/B),dim3(B,B)>>>(realResult,real_ld,testResult,test_ld,diff,N);
    
    //printMatrix_gpu(realResult,real_ld*sizeof(double),N);
    //printMatrix_gpu(testResult,test_ld*sizeof(double),N);
    
    double * diff_host = new double[N*N]();
    hipMemcpy(diff_host,diff,N*N*sizeof(double),hipMemcpyDeviceToHost);
    //  printMatrix(diff_host,N);
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            if(abs(diff_host[i*N+j])>1e-3){
                //  cout<<"diff:"<<abs(diff_host[i*N+j])<<endl;
                delete[] diff_host;
                hipFree(diff);
                return false;
            }
        }
    }
    delete[] diff_host;
    hipFree(diff);
    return true;
}


//determin the next block size
int getDynamicBlockSize(int i, int N, int B){
  float root1 = (i/448+sqrt((4*N*i-3*i*i)/448))/2;
  float root2 = ((i/448)-sqrt((4*N*i)/448-(3*i*i)/448))/2;
    
  cout<<"root1:"<<(int)root1<<"  root2:"<<(int)root2<<endl;
  if((int)root1==0){
    root1=64;
   }
  return root1+10;
}


void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,float * real_time, float * proc_time, long long * flpins, float * mflops){
  //cout<<"start my_dpotrf"<<endl;
  //initial data
  int b_size = 0;
  double * temp;
  float gemm_time =0;
  float cpu_time =0;
  //hipHostAlloc((void**)&temp,B*B*sizeof(double),hipHostMallocDefault);
  //cout<<"pinned memory initialized"<<endl;
  //intial streams----------------------------
  hipStream_t stream0;//for main loop
  hipStream_t stream1;//for dgemm part
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
 
  //cout<<"Streams initialized"<<endl;
  //intial cublas
  hipblasStatus_t hipblasStatus_t;
  hipblasHandle_t handle0;
  hipblasStatus_t = hipblasCreate(&handle0);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS NOT INITIALIZED(handle0)"<<endl;
  hipblasStatus_t = hipblasSetStream(handle0,stream0);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS SET STREAM NOT INITIALIZED(handle0)"<<endl;


  hipblasHandle_t handle1;
  hipblasStatus_t = hipblasCreate(&handle1);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS NOT INITIALIZED(handle1)"<<endl;

  hipblasStatus_t = hipblasSetStream(handle1,stream1);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS SET STREAM NOT INITIALIZED(handle1)"<<endl;

  //cout<<"cublas initialized"<<endl;
  //for timing
  hipEvent_t start0, stop0, start1, stop1;
  hipEventCreate(&start0);
  hipEventCreate(&stop0);
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  //cout<<"cuda event"<<endl;
    
  
    
  //float t=0;
  //cout<<"Inital complete"<<endl;
  
  //cout<<"entering loop"<<endl;
  //start the loop of calculation----------------------------
    
  vector<int> block_sizes =  optimalB(N, 1);


  if(PAPI_flops(real_time, proc_time, flpins, mflops)<PAPI_OK){
    cout<<"PAPI ERROR"<<endl;
    return;
  }
    
  
    
  //for(int i=0;i<N;i+=b_size){
  int i=0;
  for(int j=0;j<block_sizes.size();j++){
    i += b_size;
    b_size = block_sizes[j];
    b_size = min(b_size,N-i);

    cout<<"block size:"<<b_size<<"  ";
    
	if(i>0){
	  double alpha = -1;
	  double beta = 1;
	  //hipEventRecord(start0,stream0);
	  hipblasDsyrk(handle0,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_T,b_size,i,&alpha,matrix+i*ld,ld,&beta,matrix+i*ld+i,ld);
	  //hipEventRecord(stop0,stream0);
	}
	
    if(i!=0&&i+b_size<N){
	  double alpha = -1;
	  double beta = 1;
	  //hipEventRecord(start1,stream1);
	  hipblasDgemm(handle1,HIPBLAS_OP_T,HIPBLAS_OP_N,b_size,N-i-b_size,i,&alpha,matrix+i*ld,ld,matrix+(i+b_size)*ld,ld,&beta,matrix+(i+b_size)*ld+i,ld);
	  //hipEventRecord(stop1,stream1);
	}
      
	/*if(i>0){
	  hipEventSynchronize(stop0);
	  hipEventElapsedTime(&t,start0,stop0);
	  cout<<"SYRK: "<<t<<"ms  ";
	}*/

	
	hipStreamSynchronize(stream0);	
	//hipEventRecord(start0,stream0);
	hipHostAlloc((void**)&temp,b_size*b_size*sizeof(double),hipHostMallocDefault);
	hipMemcpy2D(temp,b_size*sizeof(double),matrix+i*ld+i,ld*sizeof(double),b_size*sizeof(double),b_size,hipMemcpyDeviceToHost);
	//hipStreamSynchronize(stream0);
    //POTF2_CPU(uplo,temp,b_size,b_size);
	int info;
    dpotrf('U',b_size,temp,b_size,&info);
	
	hipMemcpy2DAsync(matrix+i*ld+i,ld*sizeof(double),temp,b_size*sizeof(double),b_size*sizeof(double),b_size,hipMemcpyHostToDevice,stream0);
    //hipEventRecord(stop0,stream0);
	
	/*if(i!=0&&i+b_size<ld){
	  hipEventSynchronize(stop1);
	  hipEventElapsedTime(&gemm_time,start1,stop1);
	  cout<<"GEMM: "<<gemm_time<<"ms  ";
	}


    hipEventSynchronize(stop0);
	hipEventElapsedTime(&cpu_time,start0,stop0);
      cout<<"CPU: "<<cpu_time<<"ms  "<<endl;
     */
	//update B                                                                      
    if(i+b_size<N){
	  hipStreamSynchronize(stream1);
	  hipStreamSynchronize(stream0);
	  double alpha = 1;
	  
	  //hipEventRecord(start0,stream0);
	  hipblasDtrsm(handle0,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT,b_size,N-i-b_size,&alpha,matrix+i*ld+i,ld,matrix+(i+b_size)*ld+i,ld);
	  /*hipEventRecord(stop0,stream0);
	  hipEventSynchronize(stop0);
	  hipEventElapsedTime(&t,start0,stop0);
	  cout<<"TRSM: "<<t<<"ms  "<<endl;*/
    }
    hipHostFree(temp);
  }
  //  t=clock()-t;
  //  float time =((float)t/CLOCKS_PER_SEC);
  //  cout<<"time[N="<<N<<"B="<<B<<"]:"<<time<<"s."<<endl;
    
    
  if(PAPI_flops( real_time, proc_time, flpins, mflops)<PAPI_OK){
    cout<<"PAPI ERROR"<<endl;
    return;
  }
  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);
  hipblasDestroy(handle0);
  hipblasDestroy(handle1);
 // hipHostFree(temp);
  PAPI_shutdown();

}





void test_mydpotrf(int N, int B,float * real_time, float * proc_time, long long * flpins, float * mflops){

  char uplo = 'u';
  double * matrix;
  double * result;
  size_t matrix_pitch;
  size_t result_pitch;
  //Memory allocation on RAM and DRAM
  hipMallocPitch((void**)&matrix,&matrix_pitch,N*sizeof(double),N);
  hipMallocPitch((void**)&result,&result_pitch,N*sizeof(double),N);
 
  int matrix_ld= matrix_pitch/sizeof(double);
  int result_ld= result_pitch/sizeof(double);

  matrixGenerator_gpu(uplo,matrix,matrix_ld,result,result_ld,N,2);
    
  my_dpotrf(uplo,matrix,matrix_ld,N,B,real_time, proc_time, flpins, mflops);
  
  
 
  //Verify result
  if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
    cout<<"Result passed!"<<endl;
  }else{
    cout<<"Result failed!"<<endl;
  }
  
  hipFree(matrix);
  hipFree(result);
  
}

int main(int argc, char**argv){    
    
    int TEST_NUM = 1;
    int n[16]={256,384,512,640,768,896,1024,2048,3072,4096,5120,6144,7168,8192,9216,10240};
    int b=16;
    for(int k=0;k<1;k++){
        //for(int b=2;b<n;b*=2){
      //int b=OptimalB(n[k],1);
          float total_real_time = 0.0;
          float total_proc_time = 0.0;
          long long total_flpins = 0.0;
          float total_mflops = 0.0;

	  float real_time = 0.0;
	  float proc_time = 0.0;
	  long long flpins = 0.0;
	  float mflops = 0.0;
	  double flops = FLOPS_DPOTRF(n[k])/1e9;
	  cout<<"flops:"<<flops<<"  ";

          for(int i=0;i<TEST_NUM;i++){
            test_mydpotrf(n[k],b,&real_time, &proc_time, &flpins, &mflops);
            total_real_time += real_time;
            total_proc_time += proc_time;
            total_flpins += flpins;
            total_mflops += mflops;
          }
          cout<<"Size:"<<n[k]<<"("<<b<<")---Real_time:"<<total_real_time/(double)TEST_NUM<<"---"<<"Proc_time:"<<total_proc_time/(double)TEST_NUM<<"---"<<"Total GFlops:"<<flops/(total_proc_time/(double)TEST_NUM)<<endl;
         
        //}
    }


     
   /* cout<<"16Optimal:"<<OptimalB(16,1)<<endl;
    cout<<"32Optimal:"<<OptimalB(32,1)<<endl;
    cout<<"64Optimal:"<<OptimalB(64,1)<<endl;
    cout<<"128Optimal:"<<OptimalB(128,1)<<endl;
    cout<<"256Optimal:"<<OptimalB(256,1)<<endl;
    cout<<"512Optimal:"<<OptimalB(512,1)<<endl;
    cout<<"1024Optimal:"<<OptimalB(1024,10)<<endl;
    cout<<"2048Optimal:"<<OptimalB(2048,10)<<endl;
    cout<<"4096Optimal:"<<OptimalB(4096,10)<<endl;
    cout<<"8192Optimal:"<<OptimalB(8192,10)<<endl;
    cout<<"16384Optimal:"<<OptimalB(16384,100)<<endl;
    */

}
