 /*Blocked Cholesky Factorization v1.4.
 *This version mainly optimized on accessing global memory.
 *Shared memory bank conflicts are not taken care of.
 *Jieyang Chen, University of California, Riverside
 **/




#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
//#define N 1024
//#define B 64
using namespace std;

__global__ void POTF2(double * matrix, int I, int N, int B){
  int id = threadIdx.x;
  
  double * localMatrix = matrix+I*N+I;
  extern __shared__ double sharedLocalMatrix[];
  register int idB =id*B;
  //copy sub-matrix to shared memory
  for(int i=0;i<B;i++){
    sharedLocalMatrix[i*B+id] = localMatrix[i*N+id]; 
    __syncthreads();
  }

    for(int i = 0; i<B;i++){
    if(id==i){
      sharedLocalMatrix[i*B+i] = sqrt(sharedLocalMatrix[i*B+i]);
    }
    __syncthreads();
    if(id>i&&id<B){
      sharedLocalMatrix[idB+i] = sharedLocalMatrix[idB+i]/sharedLocalMatrix[i*B+i];
     sharedLocalMatrix[i*B+id] = sharedLocalMatrix[idB+i];
      __syncthreads();
      for(int j=i+1;j<B;j++){
       	sharedLocalMatrix[idB+j]-=sharedLocalMatrix[idB+i]*sharedLocalMatrix[j*B+i];
      }
    }
    __syncthreads();
  }
    __syncthreads();
    for(int i=0;i<B;i++){
      localMatrix[i*N+id]=sharedLocalMatrix[i*B+id];
      __syncthreads();
    }
}

__global__ void TRSM(double * matrix, int I,int N, int B){
  
  int id = threadIdx.x+blockIdx.x*blockDim.x;
  double * localMatrix = matrix+I*N+I;
  extern __shared__ double sharedLocalMatrix[];
  for(int i=0;i<B;i++){
      sharedLocalMatrix[i*B+threadIdx.x] = localMatrix[i*N+threadIdx.x];
      __syncthreads();
  }
  //__syncthreads();
  
  for(int row=id;row<N-I-B;row+=blockDim.x*gridDim.x){
    for(int j=0;j<B;j++){
      double sum = 0;
      for(int k=0;k<j;k++){
	sum+=localMatrix[(B+row)*N+k]*sharedLocalMatrix[k*B+j];
      }
      localMatrix[(B+row)*N+j]-=sum;
      localMatrix[(B+row)*N+j]/=sharedLocalMatrix[j*B+j];
      localMatrix[j*N+B+row] = localMatrix[(B+row)*N+j];
    }
  }
}




__global__ void RKSY(double * matrix, int I,int N, int B){
  int x = threadIdx.x+blockIdx.x*blockDim.x;
  int y = threadIdx.y+blockIdx.y*blockDim.y;
  double * localMatrix = matrix+I*N+I;
  for(int row =y;row<N-I-B;row+=blockDim.y*gridDim.y){
   for(int col = x;col<N-I-B;col+=blockDim.x*gridDim.x){
      double sum = 0;
      for(int k=0;k<B;k++){         
	sum+=localMatrix[(B+row)*N+k]*localMatrix[k*N+B+col];   
      }     
      localMatrix[(B+row)*N+B+col]-=sum;
      }
    }  
}

void CPU_CholeskyFactorization(double * matrix, int N){
    for(int i = 0; i<N;i++){ 
      matrix[i*N+i] = sqrt(matrix[i*N+i]);
      for(int j=i+1;j<N;j++){
	matrix[j*N+i] = matrix[j*N+i]/matrix[i*N+i];
	matrix[i*N+j] = matrix[j*N+i];
      }
      for(int j=i+1;j<N;j++){
	for(int k=i+1;k<N;k++){
	  matrix[j*N+k]-=matrix[j*N+i]*matrix[i*N+k];
	}
      }
  }
}



void matrixGenerator(double * matrix, double * result, int N){
  double * A = new double[N*N]();
  double * At = new double[N*N]();
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(j<i+1)
	A[i*N+j] = rand()%10+1;
      else
	A[i*N+j] = 0;
      At[j*N+i] = A[i*N+j];
    }
  }
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      for(int k=0;k<N;k++){
	matrix[i*N+j]+=A[i*N+k]*At[k*N+j];
      }
      if(matrix[i*N+j]<0)
	cout<<"Matrix generate Error!"<<endl;
    }
  }
  for(int i=0;i<N;i++){
    for(int j=i+1;j<N;j++){
      A[i*N+j] = At[i*N+j];
    }
  }
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      result[i*N+j]=A[i*N+j];
    }
  }
  free(A);
  free(At);
}

bool resultVerify(double * realResult, double * testResult, int N){
  bool pass = true;
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(realResult[i*N+j]!=testResult[i*N+j]){
	pass = false;
	break;
      }
    }
  }
  return pass;
}

void printMatrix(double * matrix, int N){
  // cout.width(5);
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cout.width(2);
      cout.setf(ios::left);   
      cout<<matrix[i*N+j];
    }
    cout<<endl;
  }
  cout<<endl;
}


void test(int N, int B){  
  double * input = new double[N*N]();
  double * result = new double[N*N]();
  matrixGenerator(input,result,N);
  double * output = new double[N*N]();
  double * dev_input;
 
  double * a =new double[N*N]();
  memcpy(a,input,N*N*sizeof(double));

  clock_t t = clock();
  CPU_CholeskyFactorization(a,N);
  t=clock()-t;
  float time =((float)t/CLOCKS_PER_SEC)*1000.0;
  cout<<"CPU time:"<<time<<"ms."<<endl;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  
  hipMalloc((void**)&dev_input, N*N*sizeof(double));
  hipMemset((void**)&dev_input,0,N*N*sizeof(double));
  hipMemcpy(dev_input,input,N*N*sizeof(double),hipMemcpyHostToDevice);
  

  //cudaEvent_t start, stop;
  //cudaEventCreate(&start);
  //cudaEventCreate(&stop);
  //cudaEventRecord(start,0);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(POTF2),hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TRSM),hipFuncCachePreferShared);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(RKSY),hipFuncCachePreferShared);
   for(int I=0;I<N;I+=B){
     POTF2<<<dim3(1),dim3(B),B*B*sizeof(double)>>>(dev_input,I,N,B);
     TRSM<<<dim3(B),dim3(B),B*B*sizeof(double)>>>(dev_input,I,N,B);
     RKSY<<<dim3(32,32),dim3(8,8)>>>(dev_input,I,N,B);
   }
   hipMemcpy(output,dev_input,N*N*sizeof(double),hipMemcpyDeviceToHost);
   hipEventRecord(stop,0);
   hipEventSynchronize(stop);
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime,start,stop);
   cout<<"Performing blocked Cholesky Factorization on GPU.Time:"<<elapsedTime<<"ms."<<endl;
  cout<<"Matrix Size:"<<N<<"*"<<N<<", Block Size:"<<B<<"*"<<B<<"."<<endl;
  //cudaMemcpy(output,dev_input,N*N*sizeof(double),cudaMemcpyDeviceToHost);
  cout<<"Verify result on CPU...";
  if(resultVerify(result,output,N)){
    cout<<"Result passed"<<endl;
  }
  else{
    cout<<"Result failed"<<endl;
  }
  hipFree(dev_input);
  free(input);
  free(output);
  // return 0;
}

int main(){
  // test(16,2);
  //test(16,4);
  //test(16,8);
  //test(32,4);
  //test(32,8);
  //test(32,16);
  //test(64,8);
  //test(64,16);
  //test(64,32);
  test(128,16);
  test(128,32);
  test(128,64);

  test(256,16);
  test(256,32);
  test(256,64);
 
  test(512,16);
  test(512,32);
  test(512,64);
  
  test(1024,16);
  test(1024,32);
  test(1024,64);

  test(2048,16);
  test(2048,32);
  test(2048,64);

  /*
  test(4096,512);
  test(4096,1024);
  test(4096,2048); 
 
  test(2048,64);
  */
}

