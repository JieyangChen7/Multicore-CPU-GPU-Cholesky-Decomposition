
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#define N 1024
#define B 64
using namespace std;

__global__ void POTF2(double * matrix, int I){
  int id = threadIdx.x;
  
  double * localMatrix = matrix+I*N+I;
  __shared__ double sharedLocalMatrix[B*B];
  register int idB =id*B;
  for(int i=0;i<B;i++){
    sharedLocalMatrix[idB+i] = localMatrix[id*N+i]; 
  }
  __syncthreads();
  for(int i = 0; i<B;i++){
    if(id==i){
      sharedLocalMatrix[i*B+i] = sqrt(sharedLocalMatrix[i*B+i]);
    }
    __syncthreads();
    if(id>i&&id<B){
      sharedLocalMatrix[idB+i] = sharedLocalMatrix[idB+i]/sharedLocalMatrix[i*B+i];
      sharedLocalMatrix[i*B+id] = sharedLocalMatrix[idB+i];
      __syncthreads();
      for(int j=i+1;j<B;j++){
       	sharedLocalMatrix[idB+j]-=sharedLocalMatrix[idB+i]*sharedLocalMatrix[j*B+i];
      }
    }
    __syncthreads();
  }
  __syncthreads();
  for(int i=0;i<B;i++){
    localMatrix[id*N+i]=sharedLocalMatrix[idB+i];
  }
  __syncthreads(); 
}


__global__ void TRSM(double * matrix, int I){
  
  int id = threadIdx.x+blockIdx.x*blockDim.x;
  double * localMatrix = matrix+I*N+I;
  __shared__ double sharedLocalMatrix[B*B];
  if(threadIdx.x<B){
    for(int i=0;i<B;i++){
      sharedLocalMatrix[threadIdx.x*B+i] = localMatrix[threadIdx.x*N+i];
    }
  }
  __syncthreads();
  
  for(int row=id;row<N-I-B;row+=blockDim.x*gridDim.x){
    for(int j=0;j<B;j++){
      double sum = 0;
      for(int k=0;k<j;k++){
	sum+=localMatrix[(B+row)*N+k]*sharedLocalMatrix[k*B+j];
      }
      localMatrix[(B+row)*N+j]-=sum;
      localMatrix[(B+row)*N+j]/=sharedLocalMatrix[j*B+j];
      localMatrix[j*N+B+row] = localMatrix[(B+row)*N+j];
    }
  }
}


__global__ void RKSY(double * matrix, int I){
  int x = threadIdx.x+blockIdx.x*blockDim.x;
  int y = threadIdx.y+blockIdx.y*blockDim.y;
  double * localMatrix = matrix+I*N+I;
  for(int row =y;row<N-I-B;row+=blockDim.y*gridDim.y){
    for(int col = x;col<N-I-B;col+=blockDim.x*gridDim.x){
      double sum = 0;
      for(int k=0;k<B;k++){         
	sum+=localMatrix[(B+row)*N+k]*localMatrix[k*N+B+col];   
      }     
      localMatrix[(B+row)*N+B+col]-=sum;
    }
  }  
}

void CPU_CholeskyFactorization(double * matrix){
  for(int i = 0; i<N;i++){ 
    matrix[i*N+i] = sqrt(matrix[i*N+i]);
    for(int j=i+1;j<N;j++){
      matrix[j*N+i] = matrix[j*N+i]/matrix[i*N+i];
      matrix[i*N+j] = matrix[j*N+i];
    }
    for(int j=i+1;j<N;j++){
      for(int k=i+1;k<N;k++){
	matrix[j*N+k]-=matrix[j*N+i]*matrix[i*N+k];
      }
    }
  }
}



void matrixGenerator(double * matrix, double * result){
  double * A = new double[N*N]();
  double * At = new double[N*N]();
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(j<i+1)
	A[i*N+j] = rand()%10+1;
      else
	A[i*N+j] = 0;
      At[j*N+i] = A[i*N+j];
    }
  }
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      for(int k=0;k<N;k++){
	matrix[i*N+j]+=A[i*N+k]*At[k*N+j];
      }
      if(matrix[i*N+j]<0)
	cout<<"Matrix generate Error!"<<endl;
    }
  }
  for(int i=0;i<N;i++){
    for(int j=i+1;j<N;j++){
      A[i*N+j] = At[i*N+j];
    }
  }
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      result[i*N+j]=A[i*N+j];
    }
  }
  free(A);
  free(At);
}

bool resultVerify(double * realResult, double * testResult){
  bool pass = true;
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      if(realResult[i*N+j]!=testResult[i*N+j]){
	pass = false;
	break;
      }
    }
  }
  return pass;
}

void printMatrix(double * matrix){
  // cout.width(5);
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
      cout.width(2);                                                                                                                        
      cout.setf(ios::left);   
      cout<<matrix[i*N+j];
    }
    cout<<endl;
  }
  cout<<endl;
}


int main(){  
  double * input = new double[N*N]();
  double * result = new double[N*N]();
  matrixGenerator(input,result);
  double * output = new double[N*N]();
  double * dev_input;
  
  double * a =new double[N*N]();
  memcpy(a,input,N*N*sizeof(double));
  
  clock_t t = clock();
  CPU_CholeskyFactorization(a);
  t=clock()-t;
  float time =((float)t/CLOCKS_PER_SEC)*1000.0;
  cout<<"CPU time:"<<time<<"ms."<<endl;
  
  hipMalloc((void**)&dev_input, N*N*sizeof(double));
  hipMemset((void**)&dev_input,0,N*N*sizeof(double));
  hipMemcpy(dev_input,input,N*N*sizeof(double),hipMemcpyHostToDevice);
  
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  
 
 

  for(int I=0;I<N;I+=B){
 hipFuncSetCacheConfig(reinterpret_cast<const void*>(POTF2), hipFuncCachePreferShared); 
   POTF2<<<dim3(1),dim3(B)>>>(dev_input,I);
   hipDeviceSynchronize();
 hipFuncSetCacheConfig(reinterpret_cast<const void*>(TRSM), hipFuncCachePreferShared);
   TRSM<<<dim3(1),dim3(B)>>>(dev_input,I);
 hipDeviceSynchronize();
 hipFuncSetCacheConfig(reinterpret_cast<const void*>(RKSY), hipFuncCachePreferShared);
    RKSY<<<dim3(32,32),dim3(8,8)>>>(dev_input,I);
 hipDeviceSynchronize();
  }
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,start,stop);
  cout<<"Performing blocked Cholesky Factorization on GPU.Time:"<<elapsedTime<<"ms."<<endl;
  cout<<"Matrix Size:"<<N<<"*"<<N<<", Block Size:"<<B<<"*"<<B<<"."<<endl;
  hipMemcpy(output,dev_input,N*N*sizeof(double),hipMemcpyDeviceToHost);
  cout<<"Verify result on CPU...";
  if(resultVerify(result,output)){
    cout<<"Result passed"<<endl;
  }
  else{
    cout<<"Result failed"<<endl;
  }
  hipFree(dev_input);
  free(input);
  free(output);
  return 0;
}
/*
  int main(){
  test(16,2);
  test(16,4);
  test(16,8);
  test(32,4);
  test(32,8);
  test(32,16);
  test(64,8);
  test(64,16);
  test(64,32);
  test(128,16);
  test(128,32);
  test(128,64);
  test(256,32);
  test(256,64);
  test(256,128);
  test(512,64);
  test(512,128);
  test(512,256);
  
  test(1024,128);
  test(1024,256);
  test(1024,512);
  test(2048,256);
  test(2048,512);
  test(2048,1024);

  
  test(4096,512);
  test(4096,1024);
  test(4096,2048); 
  
}
*/
