#include "hip/hip_runtime.h"
/*Blocked Cholesky Factorization v1.4.
*potf on CPU and dtrsm on GPU, dgemm on GPU. Compute either upper or lower. Initial data is on GPU, so transfer the data to GPU is not taken care of.
*Jieyang Chen, University of California, Riverside
**/

//Initial Data on GPU
//Hybird GPU (DTRSM & DGEMM)and CPU (DPOTRF) version MAGMA way
//Column Major
//Either upper and lower triangle
//testing function are made to facilitate testing
//CPU and GPU are asynchronized
//CUBLAS are used in DTRSM & DGEMM
//Leading Dimension is used
//Add CUDA Event timing

#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath> 
#include<ctime>
#include"hipblas.h"
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include"acml.h"
#include"papi.h"
#include"cblas.h"
//#include"lapacke.h"
//#include"blas.h"
//#include<hip/hip_runtime.h>
//#include<hip/hip_runtime.h>

#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;

void printMatrix_host(double * matrix_host, int N);
void printMatrix_gpu(double * matrix_device, size_t matrix_pitch, int N);
void POTF2_CPU(char uplo, double * matrix, int ld, int B);
__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta);
void matrixGenerator_gpu(char uplo, double * matrix, int matrix_ld, double * result, int result_ld, int N,  int B);
__global__ void resultVerify_gpu_help(double * realResult,int real_ld, double * testResult,int test_ld,double * diff, int N);
bool resultVerify_gpu(double * realResult,int real_ld, double * testResult, int test_ld, int N, int B);
void my_dpotrf(char uplo, double * matrix, int ld, int N, int B, double k, float * real_time, float * proc_time, long long * flpins, float * mflops);


long long int SYRK_Flops(long long N, long long B, long long I){
    long long int flop_num = 0;
    //int b_size;
    //for(int i=B;i<N;i+=B){
        //b_size = min(B,N-i);
        //flop_num += 2 * b_size * b_size * i + b_size*b_size;
    
    //}
    //return flop_num;
    if (N>0&&(!(I<B))&&I<N) {
        
        flop_num = 2 * B * B * I + B*B;
        if (flop_num<0) {
            cout<<"ERROR:syrk"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}

long long GEMM_Flops(long long N, long long B, long long I){
    long long flop_num = 0;
    /*int b_size;
    for(int i=B;i<N-B;i+=B){
        b_size = min(B,N-i);
        flop_num += 2 * b_size * (N-i-b_size) * i + b_size*(N-i-b_size);
    }
    return flop_num;
    */
    if (N>0&&(!(I<B))&&N-I-B>0) {
        flop_num = 2 * B * (N-I-B) * I + B*(N-I-B);
        if (flop_num<0) {
            cout<<"ERROR:gemm"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}
        
long long TRSM_Flops(long long N, long long B, long long I){
    long long flop_num = 0;
    /*int b_size;
    for(int i=0;i<N-B;i+=B){
        b_size = min(B,N-i);
        flop_num += b_size * b_size * (N-i-b_size);
    }
    return flop_num;
     */
    if (N>0&&(!(I<0))&&N-I-B>0) {
        flop_num = B * B * (N-I-B);
        if (flop_num<0) {
            cout<<"ERROR:trsm"<<flop_num<<"  "<<N<<" "<<B<<" "<<I<<endl;
        }
        return flop_num;
    }
    return 0;
}

long POTRF_Flops(int B){
    long long flop_num = 0;
    int b_size = B;
    if(B>64){
        for (int i=0; i<B; i+=64) {
            b_size = min(64, B-i);
            //SYRK
            if(i>0){
                flop_num += SYRK_Flops(B, b_size, i);
            }
            //GEMM
            if(i>0&&i<B-b_size){
                flop_num += GEMM_Flops(B, b_size, i);
            }
            //POTRF2
            for(int j=0;j<b_size;j++){
                flop_num += 2*j + 1 + (b_size-j-1)*i + b_size-j-1;
            }
            //TRSM
            flop_num += TRSM_Flops(B, b_size, i);
        }
    }else{
      for(int i=0;i<B;i++){
          flop_num += 2*i + 1 + (B-i-1)*i + B-i-1;
      }
    }
    if (flop_num<0) {
        cout<<"ERROR:potrf  "<<flop_num<<"  "<<B<<" "<<endl;
    }
    return flop_num;
}

double COPY_time(int B){
    
    float real_time = 0.0;
    float proc_time = 0.0;
    long long flpins = 0.0;
    float mflops = 0.0;
    
    
    float total_real_time = 0.0;
    float total_proc_time = 0.0;
    long long total_flpins = 0.0;
    float total_mflops = 0.0;
    
    int TEST_NUM = 10;
    
    char uplo = 'u';
    double * matrix;
    double * result;
    double * temp;
    
    size_t matrix_pitch;
    size_t result_pitch;
    //Memory allocation on RAM and DRAM
    hipMallocPitch((void**)&matrix,&matrix_pitch,B*sizeof(double),B);
    hipMallocPitch((void**)&result,&result_pitch,B*sizeof(double),B);
    hipHostAlloc((void**)&temp,B*B*sizeof(double),hipHostMallocDefault);
    
    int matrix_ld= matrix_pitch/sizeof(double);
    int result_ld= result_pitch/sizeof(double);
    
    matrixGenerator_gpu(uplo,matrix,matrix_ld,result,result_ld,B,2);
    
    
    for(int i=0;i<TEST_NUM;i++){
    
      if(PAPI_flops( &real_time, &proc_time, &flpins, &mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return -1;
      }
      hipMemcpy2D(temp,B*sizeof(double),matrix,matrix_pitch,B*sizeof(double),B,hipMemcpyDeviceToHost);
      hipMemcpy2D(matrix,matrix_pitch,temp,B*sizeof(double),B*sizeof(double),B,hipMemcpyHostToDevice);
      if(PAPI_flops( &real_time, &proc_time, &flpins, &mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return -1;
      }
      PAPI_shutdown();
      total_real_time += real_time;
      total_proc_time += proc_time;
    }
    

    
    
    
    hipHostFree(temp);
    hipFree(matrix);
    hipFree(result);
    
    double time =(total_real_time/(double)TEST_NUM);
    if (time<0) {
        cout<<"ERROR:copy"<<B<<" "<<endl;
    }
    return time;

}

int OptimalB(int N, int G){
    long long GPUflops = 300000000000;
    long long CPUflops = 9600000000;
    int bestB = 2;
    double bestT = 10000000;
    long long numc=0;
    long long numg=0;
    long long bnumc=0;
    long long bnumg=0;
    /*
    for(int b=2;b<N;b++){ cout<<"b="<<b;
        double syrk_time = SYRK_Flops(N,b)/GPUflops; cout<<"---syrk_time="<<syrk_time;
        double gemm_time = GEMM_Flops(N,b)/GPUflops;cout<<"---gemm_time="<<gemm_time;
        double trsm_time = TRSM_Flops(N,b)/GPUflops;cout<<"---trsm_time="<<trsm_time;
        double potrf_time = POTRF_Flops(N,b)/CPUflops; cout<<"---potrf_time="<<potrf_time;
        double copy_time = COPY_time(N,b); cout<<"---copy_time="<<copy_time;
        
        double total_time = syrk_time + trsm_time + max(gemm_time,potrf_time+copy_time); cout<<"---total_time="<<total_time<<endl;
        
        if(total_time<bestT){
            bestT = total_time;
            bestB = b;
        }
        
    }
     */
    //    long long num256 =0;
    for (int B=2; B<N+1; B+=G) {
      numc=0;
      numg=0;
        double current_time = 0;
        double copy_t = (double)COPY_time(B);
        double potrf_t = (double)POTRF_Flops(B)/CPUflops;
	numc+= POTRF_Flops(B)*N/B;
        for (int i=0; i<N; i+=B) {
	  if (i>0) {
	    numg+=SYRK_Flops(N,B,i);
	    current_time += (double)SYRK_Flops(N,B,i)/GPUflops;
	  }
	  if(i>0 && i<N-B){
	    numg+=GEMM_Flops(N,B,i);
	    current_time += (double)GEMM_Flops(N,B,i)/GPUflops;
	  }
            double t1 = 0;
            double t2 = 0;
            t1 += potrf_t;
            t1 += copy_t;
            if(i>0 && i<N-B){
	      numg+=GEMM_Flops(N,B,i);
	      t2 += (double)GEMM_Flops(N,B,i)/GPUflops;
            }
            current_time += max(t1,t2);
            if(i<N-B){
	      numg+=TRSM_Flops(N,B,i);
	      current_time += (double)TRSM_Flops(N,B,i)/GPUflops;
            }
        }
	if(B==256){
	  cout<<"num256:"<<numc<<"  "<<numg<<"   "<<current_time<<endl;
	}
        if(current_time < bestT){
	  bestT = current_time;
            bestB = B;
	    bnumc = numc;
	    bnumg = numg;
           
        }
    }
    cout<<"N"<<N<<"--numc:"<<bnumc<<"--numg"<<bnumg<<"   b:"<<bestB<<"time:"<<bestT<<endl;
    
    return bestB;
    
    
    
}


void printMatrix_host(double * matrix_host, int N){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            cout.width(5);
            cout.setf(ios::left);
            cout<<matrix_host[j*N+i];
        }
        cout<<endl;
    }
    cout<<endl;
}

void printMatrix_gpu(double * matrix_device, size_t matrix_pitch, int N){
    double * matrix_host = new double[N*N]();
    hipMemcpy2D(matrix_host,N*sizeof(double),matrix_device,matrix_pitch,N*sizeof(double),N,hipMemcpyDeviceToHost);
    printMatrix_host(matrix_host,N);
    delete[] matrix_host;
}

//cpu version
void POTF2_CPU(char uplo, double * matrix, int ld, int B){
  if(uplo == 'u'){
    for(int i = 0; i<B;i++){
      matrix[i*ld+i] = sqrt(matrix[i*ld+i]);
      for(int j=i+1;j<B;j++){
        matrix[j*ld+i] /=matrix[i*ld+i];
      }
      for(int j=i+1;j<B;j++){
        for(int k=i+1;k<j+1;k++){
          matrix[j*ld+k]-=matrix[j*ld+i]*matrix[k*ld+i];
        }
      }
    }
  } 
  if(uplo == 'l'){
    for(int i = 0; i<B;i++){
      matrix[i*ld+i] = sqrt(matrix[i*ld+i]);
      for(int j=i+1;j<B;j++){
	matrix[i*ld+j] /=matrix[i*ld+i];
      }
      for(int j=i+1;j<B;j++){
	for(int k=i+1;k<j+1;k++){
	  matrix[k*ld+j]-=matrix[i*ld+j]*matrix[i*ld+k];
	}
      }
    }
  }
}



__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta){
    int col = threadIdx.x+blockIdx.x*blockDim.x;
    int row = threadIdx.y+blockIdx.y*blockDim.y;
    if(uplo == 'u'){
        if(row<col+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    }
    else{
        if(col<row+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    } 
}


void matrixGenerator_gpu(char uplo, double * matrix, int matrix_ld, double * result, int result_ld, int N,  int B){
    double a = 10.0;
    //initialize cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    //initialize hiprand
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,10ULL);
    //generate random number in range (0,1] on result using hiprand
    hiprandGenerateUniformDouble(gen,result,result_ld*N);
    hipDeviceSynchronize();
    //printMatrix_gpu(result,result_ld*sizeof(double),N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(result, result_ld, uplo, a,1);
    hipDeviceSynchronize();
    //printMatrix_gpu(result,N);
    //do matrix-matrix multiplcation using cublas
    hipMemset(matrix,0,matrix_ld*N*sizeof(double));
    
    
    double alpha = 1.0;
    double beta = 1.0;
    if(uplo == 'u'){
        hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,N,N,N,&alpha,result,result_ld,result,result_ld,&beta,matrix,matrix_ld);
    }
    else if(uplo == 'l'){
        hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,N,N,&alpha,result,result_ld,result,result_ld,&beta,matrix,matrix_ld);
    }
    hipDeviceSynchronize();
    //printMatrix_gpu(matrix,N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(matrix, matrix_ld, uplo, 1.0,0);
    hipDeviceSynchronize();
    //printMatrix_gpu(matrix,matrix_ld*sizeof(double), N);
}
 

__global__ void resultVerify_gpu_help(double * realResult,int real_ld, double * testResult,int test_ld,double * diff, int N){
    int col = threadIdx.x+blockIdx.x*blockDim.x;
    int row = threadIdx.y+blockIdx.y*blockDim.y;
    diff[col*N+row] = testResult[col*test_ld+row] - realResult[col*real_ld+row];
}

bool resultVerify_gpu(double * realResult,int real_ld, double * testResult, int test_ld, int N, int B){
    double * diff;
    hipMalloc((void**)&diff,N*N*sizeof(double));
    resultVerify_gpu_help<<<dim3(N/B,N/B),dim3(B,B)>>>(realResult,real_ld,testResult,test_ld,diff,N);
    
    //printMatrix_gpu(realResult,real_ld*sizeof(double),N);
    //printMatrix_gpu(testResult,test_ld*sizeof(double),N);
    
    double * diff_host = new double[N*N]();
    hipMemcpy(diff_host,diff,N*N*sizeof(double),hipMemcpyDeviceToHost);
    //  printMatrix(diff_host,N);
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            if(abs(diff_host[i*N+j])>1e-3){
                //  cout<<"diff:"<<abs(diff_host[i*N+j])<<endl;
                delete[] diff_host;
                hipFree(diff);
                return false;
            }
        }
    }
    delete[] diff_host;
    hipFree(diff);
    return true;
    
    
    
}






//determin the next block size
int getDynamicBlockSize(int i, int N, int B){
  float root1 = (i/448+sqrt((4*N*i-3*i*i)/448))/2;
  float root2 = ((i/448)-sqrt((4*N*i)/448-(3*i*i)/448))/2;
    
  cout<<"root1:"<<(int)root1<<"  root2:"<<(int)root2<<endl;
  if((int)root1==0){
    root1=64;
   }
  return root1+10;


}


void my_dpotrf(char uplo, double * matrix, int ld, int N, int B,double k,float * real_time, float * proc_time, long long * flpins, float * mflops){
  //cout<<"start my_dpotrf"<<endl;
  //initial data
  int b_size = B;
  double * temp;
  float gemm_time =0;
  float cpu_time =0;
  //hipHostAlloc((void**)&temp,B*B*sizeof(double),hipHostMallocDefault);
  //cout<<"pinned memory initialized"<<endl;
  //intial streams----------------------------
  hipStream_t stream0;//for main loop
  hipStream_t stream1;//for dgemm part
  hipStreamCreate(&stream0);
  hipStreamCreate(&stream1);
 
  //cout<<"Streams initialized"<<endl;
  //intial cublas
  hipblasStatus_t hipblasStatus_t;
  hipblasHandle_t handle0;
  hipblasStatus_t = hipblasCreate(&handle0);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS NOT INITIALIZED(handle0)"<<endl;
  hipblasStatus_t = hipblasSetStream(handle0,stream0);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS SET STREAM NOT INITIALIZED(handle0)"<<endl;


  hipblasHandle_t handle1;
  hipblasStatus_t = hipblasCreate(&handle1);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS NOT INITIALIZED(handle1)"<<endl;

  hipblasStatus_t = hipblasSetStream(handle1,stream1);
  if(hipblasStatus_t != HIPBLAS_STATUS_SUCCESS)
    cout<<"CUBLAS SET STREAM NOT INITIALIZED(handle1)"<<endl;

  //cout<<"cublas initialized"<<endl;
  //for timing
  hipEvent_t start0, stop0, start1, stop1;
  hipEventCreate(&start0);
  hipEventCreate(&stop0);
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  //cout<<"cuda event"<<endl;
    
  
    
  //float t=0;
  //cout<<"Inital complete"<<endl;
  
  //cout<<"entering loop"<<endl;
  //start the loop of calculation----------------------------
    
  if(PAPI_flops(real_time, proc_time, flpins, mflops)<PAPI_OK){
    cout<<"PAPI ERROR"<<endl;
    return;
  }
    
   // clock_t t = clock();
    
  for(int i=0;i<N;i+=B){
    /*  if(i!=0){
      if(cpu_time<gemm_time){
          B+=150;
      }
      else{
          B-=150;
      }
      }
    */
        b_size = min(B,N-i);
    //cout<<"block size:"<<b_size<<"  ";
	
	if(i>0){
	  double alpha = -1;
	  double beta = 1;
	  //hipEventRecord(start0,stream0);
	  hipblasDsyrk(handle0,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_T,b_size,i,&alpha,matrix+i*ld,ld,&beta,matrix+i*ld+i,ld);
	  //hipEventRecord(stop0,stream0);
	}
	
    if(i!=0&&i+b_size<N){
	  double alpha = -1;
	  double beta = 1;
	  //int l = (N-i-b_size)*k;
	  //hipEventRecord(start1,stream1);
	  hipblasDgemm(handle1,HIPBLAS_OP_T,HIPBLAS_OP_N,b_size,N-i-b_size,i,&alpha,matrix+i*ld,ld,matrix+(i+b_size)*ld,ld,&beta,matrix+(i+b_size)*ld+i,ld);
	  //hipEventRecord(stop1    ,stream1);
	  /*if(l>0){
            double * temp2;
            hipHostAlloc((void**)&temp2,i*b_size*sizeof(double),hipHostMallocDefault);
	    double * temp3;
            hipHostAlloc((void**)&temp3,i*l*sizeof(double),hipHostMallocDefault);

            hipMemcpy2DAsync(temp2,i*sizeof(double),matrix+i*ld,ld*sizeof(double),i*sizeof(double),b_size,hipMemcpyDeviceToHost,stream0);
            hipMemcpy2DAsync(temp3,i*sizeof(double),matrix+(i+b_size+(N-i-b_size-l))*ld,ld*sizeof(double),i*sizeof(double),l,hipMemcpyDeviceToHost,stream0);
	    hipStreamSynchronize(stream0);
            dgemm('t','n',b_size,l,i,-1,temp2,i,temp3,i,1,temp3,i);
            hipMemcpy2DAsync(matrix+(i+b_size+(N-i-b_size-l))*ld,ld*sizeof(double),temp3,i*sizeof(double),i*sizeof(double),l,hipMemcpyHostToDevice,stream0);
            hipStreamSynchronize(stream0);
            hipHostFree(temp2);
	    hipHostFree(temp3);
          }
	  */
    }
      
	/*if(i>0){
	  hipEventSynchronize(stop0);
	  hipEventElapsedTime(&t,start0,stop0);
	  cout<<"SYRK: "<<t<<"ms  ";
	}*/

	
	hipStreamSynchronize(stream0);	
	//hipEventRecord(start0,stream0);
	hipHostAlloc((void**)&temp,b_size*b_size*sizeof(double),hipHostMallocDefault);
	hipMemcpy2DAsync(temp,b_size*sizeof(double),matrix+i*ld+i,ld*sizeof(double),b_size*sizeof(double),b_size,hipMemcpyDeviceToHost,stream0);
	hipStreamSynchronize(stream0);
    //POTF2_CPU(uplo,temp,b_size,b_size);
	int info;
    dpotrf('U',b_size,temp,b_size,&info);
	
	hipMemcpy2DAsync(matrix+i*ld+i,ld*sizeof(double),temp,b_size*sizeof(double),b_size*sizeof(double),b_size,hipMemcpyHostToDevice,stream0);
    //hipEventRecord(stop0,stream0);
	
	/*if(i!=0&&i+b_size<ld){
	  hipEventSynchronize(stop1);
	  hipEventElapsedTime(&gemm_time,start1,stop1);
	  cout<<"GEMM: "<<gemm_time<<"ms  ";
	}


    hipEventSynchronize(stop0);
	hipEventElapsedTime(&cpu_time,start0,stop0);
      cout<<"CPU: "<<cpu_time<<"ms  "<<endl;
     */
	//update B                                                                      
    if(i+b_size<N){
	  hipStreamSynchronize(stream1);
	  hipStreamSynchronize(stream0);
	  double alpha = 1;
          int l = (N-i-b_size)*k; 
	  //hipEventRecord(start0,stream0);
	  hipblasDtrsm(handle0,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,HIPBLAS_OP_T,HIPBLAS_DIAG_NON_UNIT,b_size,N-i-b_size-l,&alpha,matrix+i*ld+i,ld,matrix+(i+b_size)*ld+i,ld);
	 
	  if(l>0){
	    double * temp2;
	    hipHostAlloc((void**)&temp2,b_size*l*sizeof(double),hipHostMallocDefault);  
	    hipMemcpy2DAsync(temp2,b_size*sizeof(double),matrix+(i+b_size+(N-i-b_size-l))*ld+i,ld*sizeof(double),b_size*sizeof(double),l,hipMemcpyDeviceToHost,stream1);
	    hipStreamSynchronize(stream1);
	    dtrsm('l','u','t','n',B,l,1,temp,b_size,temp2,B);
	    hipMemcpy2DAsync(matrix+(i+b_size+(N-i-b_size-l))*ld+i,ld*sizeof(double),temp2,b_size*sizeof(double),b_size*sizeof(double),l,hipMemcpyHostToDevice,stream1);
	    hipStreamSynchronize(stream1);
	    hipHostFree(temp2);
	  }
	  /*hipEventRecord(stop0,stream0);
	  hipEventSynchronize(stop0);
	  hipEventElapsedTime(&t,start0,stop0);
	  cout<<"TRSM: "<<t<<"ms  "<<endl;*/
    }
    hipHostFree(temp);
  }
  //  t=clock()-t;
  //  float time =((float)t/CLOCKS_PER_SEC);
  //  cout<<"time[N="<<N<<"B="<<B<<"]:"<<time<<"s."<<endl;
    
    
  if(PAPI_flops( real_time, proc_time, flpins, mflops)<PAPI_OK){
    cout<<"PAPI ERROR"<<endl;
    return;
  }
  hipStreamSynchronize(stream0);
  hipStreamSynchronize(stream1);
  hipblasDestroy(handle0);
  hipblasDestroy(handle1);
 // hipHostFree(temp);
  PAPI_shutdown();

}





void test_mydpotrf(int N, int B,double k,float * real_time, float * proc_time, long long * flpins, float * mflops){

  char uplo = 'u';
  double * matrix;
  double * result;
  size_t matrix_pitch;
  size_t result_pitch;
  //Memory allocation on RAM and DRAM
  hipMallocPitch((void**)&matrix,&matrix_pitch,N*sizeof(double),N);
  hipMallocPitch((void**)&result,&result_pitch,N*sizeof(double),N);
 
  int matrix_ld= matrix_pitch/sizeof(double);
  int result_ld= result_pitch/sizeof(double);

  matrixGenerator_gpu(uplo,matrix,matrix_ld,result,result_ld,N,2);
    
  my_dpotrf(uplo,matrix,matrix_ld,N,B,k,real_time, proc_time, flpins, mflops);
  
  
 
  //Verify result
  if(resultVerify_gpu(result,result_ld,matrix,matrix_ld,N,2)){
    cout<<"Result passed!"<<endl;
  }else{
    cout<<"Result failed!"<<endl;
   }
  
    hipFree(matrix);
    hipFree(result);

}

int main(int argc, char**argv){    
    
    int TEST_NUM = 1;
    int n[16]={256,384,512,640,768,896,1024,2048,3072,4096,5120,6144,7168,8192,9216,10240};
    //int b[16]={88,96,104,80,154,112,86,256,512,512,732,1024,1792,2048,3072,3414};
    //int b=256;
    for(int k=0;k<16;k++){
        //for(int b=2;b<n;b*=2){
      for(double K=0.1;K<0.2;K+=0.1){
	int b=OptimalB(n[k],1)
          float total_real_time = 0.0;
          float total_proc_time = 0.0;
          long long total_flpins = 0.0;
          float total_mflops = 0.0;

	  float real_time = 0.0;
	  float proc_time = 0.0;
	  long long flpins = 0.0;
	  float mflops = 0.0;
	  double flops = FLOPS_DPOTRF(n[k])/1e9;
	  cout<<"flops:"<<flops<<"  ";

          for(int i=0;i<TEST_NUM;i++){
            test_mydpotrf(n[k],b,K,&real_time, &proc_time, &flpins, &mflops);
            total_real_time += real_time;
            total_proc_time += proc_time;
            total_flpins += flpins;
            total_mflops += mflops;
          }
      
          cout<<"Size:"<<n[k]<<"("<<b[k]<<")---Real_time:"<<total_real_time/(double)TEST_NUM<<"---"<<"Proc_time:"<<total_proc_time/(double)TEST_NUM<<"---"<<"Total GFlops:"<<flops/(total_proc_time/(double)TEST_NUM)<<endl;
      }
        //}
    }


     
  //cout<<OptimalB(16,1)<<endl;
    //cout<<OptimalB(32,1)<<endl;
    // cout<<OptimalB(64,1)<<endl;
    //    cout<<OptimalB(128,1)<<endl;

  //    cout<<OptimalB(256,1)<<endl;
    //cout<<OptimalB(384,1)<<endl;
    //cout<<OptimalB(512,1)<<endl;
    /*
    cout<<OptimalB(640,1)<<endl;
    cout<<OptimalB(768,1)<<endl;
    cout<<OptimalB(896,1)<<endl;
    cout<<OptimalB(1024,1)<<endl;
    cout<<OptimalB(2048,1)<<endl;
    cout<<OptimalB(3072,1)<<endl;
    cout<<OptimalB(4096,1)<<endl;
    cout<<OptimalB(5120,1)<<endl;
    cout<<OptimalB(6144,1)<<endl;
    cout<<OptimalB(7168,1)<<endl;
    cout<<OptimalB(8192,1)<<endl;
    cout<<OptimalB(9216,1)<<endl;
    cout<<OptimalB(10240,1)<<endl;
    */

}
