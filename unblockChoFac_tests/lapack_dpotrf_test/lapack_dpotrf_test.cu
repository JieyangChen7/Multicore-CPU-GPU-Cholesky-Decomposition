#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<iomanip>
#include<cmath>
#include<ctime>
#include"cblas.h"
#include<hiprand.h>
#include"hipblas.h"
#include "lapacke.h"
#include "blas.h"
#include "papi.h"


#define FMULS_POTRF(__n) ((__n) * (((1. / 6.) * (__n) + 0.5) * (__n) + (1. / 3.)))
#define FADDS_POTRF(__n) ((__n) * (((1. / 6.) * (__n)      ) * (__n) - (1. / 6.)))
#define FLOPS_DPOTRF(__n) (FMULS_POTRF((double)(__n))+FADDS_POTRF((double)(__n)) )

using namespace std;

void matrixGenerator_gpu(char uplo, double * matrix, double * result, int N, int B);
__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta);

void test(int c, int N, int B, char uplo, float * real_time, float * proc_time, long long * flpins, float * mflops){
    double * matrix;
    double * result;
    double * temp = new double[N*N]();
    int info = 0;
    hipMalloc((void**)&matrix,N*N*sizeof(double));
    hipMalloc((void**)&result,N*N*sizeof(double));
    
    matrixGenerator_gpu(uplo, matrix, result, N, B);

    /*  if(PAPI_flops(real_time, proc_time, flpins, mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return;
    }
    */

    hipMemcpy2D(temp,
                 N*sizeof(double),
                 matrix,
                 N*sizeof(double),
                 N*sizeof(double),
                 N,
                 hipMemcpyDeviceToHost);
    if(PAPI_flops(real_time, proc_time, flpins, mflops)<PAPI_OK){
      cout<<"PAPI ERROR"<<endl;
      return;
    }
    LAPACK_dpotrf(&uplo,&N,temp,&N,&info);
    if(PAPI_flops(real_time, proc_time, flpins, mflops)<PAPI_OK){
      cout<<"PAPI ERROR"<<endl;
      return;
    }
    hipMemcpy2D(matrix,
                 N*sizeof(double),
                 temp,
                 N*sizeof(double),
                 N*sizeof(double),
                 N,
                 hipMemcpyHostToDevice);
    /*
    if(PAPI_flops( real_time, proc_time, flpins, mflops)<PAPI_OK){
        cout<<"PAPI ERROR"<<endl;
        return;
    }
    */

    delete[] temp;
    hipFree(matrix);
    hipFree(result);
    PAPI_shutdown();
}

int main(){
    float real_time = 0.0;
    float proc_time = 0.0;
    long long flpins = 0.0;
    float mflops = 0.0;
    
    float total_real_time = 0.0;
    float total_proc_time = 0.0;
    long long total_flpins = 0.0;
    float total_mflops = 0.0;
    
    int TEST_NUM = 1000;
    
    for(int n=2;n<1025;n*=2){
        for(int i=0;i<TEST_NUM;i++){
            test(0,n,2,'u',&real_time,&proc_time,&flpins,&mflops);
            total_real_time += real_time;
            total_proc_time += proc_time;
            total_flpins += flpins;
            total_mflops += mflops;
        }
        cout<<"LAPACK:";
        cout<<"Size:"<<n<<"---Real_time:"<<total_real_time/(double)TEST_NUM<<"---"<<"Proc_time:"<<total_proc_time/(double)TEST_NUM<<"---"<<"Total flpins:"<<total_flpins/(double)TEST_NUM<<"---"<<"MFLOPS:"<<total_mflops/(double)TEST_NUM<<"GFlops:"<<FLOPS_DPOTRF(n)/(total_proc_time/(double)TEST_NUM)<<endl;
    }
    
}


void matrixGenerator_gpu(char uplo, double * matrix, double * result, int N, int B){
    double a = 10.0;
    //initialize cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    //initialize hiprand
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen,10ULL);
    //generate random number in range (0,1] on result using hiprand
    hiprandGenerateUniformDouble(gen,result,N*N);
    hipDeviceSynchronize();
    //  printMatrix_gpu(result,N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(result, N, uplo, a,1);
    hipDeviceSynchronize();
    //printMatrix_gpu(result,N);
    //do matrix-matrix multiplcation using cublas
    hipMemset(matrix,0,N*N*sizeof(double));
    
    
    double alpha = 1.0;
    double beta = 1.0;
    if(uplo == 'u'){
        hipblasDgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,N,N,N,&alpha,result,N,result,N,&beta,matrix,N);
    }
    else if(uplo == 'l'){
        hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,N,N,&alpha,result,N,result,N,&beta,matrix,N);
    }
    hipDeviceSynchronize();
    //    printMatrix_gpu(matrix,N);
    matrixDiagonalizeAndScale<<<dim3(N/B,N/B),dim3(B,B)>>>(matrix, N, uplo, 1.0,0);
    hipDeviceSynchronize();
    // printMatrix_gpu(matrix,N);
}

__global__ void matrixDiagonalizeAndScale(double * matrix, int ld, char uplo, double alpha, double beta){
    int col = threadIdx.x+blockIdx.x*blockDim.x;
    int row = threadIdx.y+blockIdx.y*blockDim.y;
    if(uplo == 'u'){
        if(row<col+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    }
    else{
        if(col<row+1){
            matrix[col*ld+row] = int(matrix[col*ld+row]*alpha+beta);
        }
        else{
            matrix[col*ld+row] = int(0.0);
        }
    } 
}
